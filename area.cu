
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 1280000
#define THREADS 128

__device__ float f(float x)
{
    // (1) COMPLETE f(x) �Լ��� ����
   float temp;
   temp = x*x+2; 
   return x/(temp*temp*temp);
}

__global__ void area_kernel(float *sums)
{
    float a, b, dx, x, y;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ float sdata[THREADS];

    // (2) COMPLETE: a,b������ ���̸� ����Ѵ�.
    a = 0.0;
    b = 2.0;
    dx = (b - a) / N;

    x = a  + i * dx;
    y = f(x);

    // ����������� ���� ���Ѵ�.
    sdata[threadIdx.x] = 0.5 * dx * (y + f(x + dx));

    // Synchronize threads before reduction
    __syncthreads();

    // doreduce in shared memory
    for (unsigned int s=blockDim.x/2; s>0; s>>=1)
    {
        if (threadIdx.x < s)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + s];
            __syncthreads();
        }
    }

    // write result for this block to global memory
    if (threadIdx.x == 0)
    {
        sums[blockIdx.x] = sdata[0];
    }
}

int main()
{
   float *sums, *sums_d, area, elapsed;
   int i;
    
   dim3 dimBlock(THREADS);
   dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x);
   hipEvent_t start, stop;
   
   hipMalloc((void **)&sums_d, sizeof(float) * dimGrid.x);
   sums = (float *)malloc(sizeof(float) * dimGrid.x);
    

   // (3) COMPLETE (timing -start)
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start, 0);

   // (4) COMPLETE: call GPU function
   area_kernel<<<dimGrid, dimBlock>>>(sums_d);

   // Copy values from GPU memory to CPU memory
   hipMemcpy(sums, sums_d, sizeof(float) * dimGrid.x, hipMemcpyDeviceToHost);
    
   // (5) COMPLETE: timing -stop
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&elapsed, start, stop);
    
   hipEventDestroy(start);
   hipEventDestroy(stop);

   // add the computed value to the value of pi
   area = 0.0;
   for (i = 0; i < dimGrid.x; i++)
      area += sums[i];

   printf("area: %5.10f\n", area);
   printf("elapsed time: %f milliseconds\n", elapsed);
   printf("GFLOPS: %5.2f\n", (N * 12.0 / (elapsed / 1000.0)) / 1000000000.0); // I changed #Flops 16 to 12, because my prgram's #flops is 12.

   hipFree(sums_d);
   free(sums);
    
   return 0;
}
