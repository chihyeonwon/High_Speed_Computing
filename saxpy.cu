
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 10000
#define THREADS 100

__global__ void saxpy(float *A, float *B, float X, float Y) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
}

int main() {
   float A[N], B[N], B2[N], X, Y;
   float *d_A, *d_B;  // Device pointer for B2
   int i;
    
   // Initialize A on the host (CPU)
   for (int i = 0; i < N; i++)
      A[i] = i * 2;

   X = 1.23;
   Y = 2.34;
    
   hipSetDevice(1); //cs17 17�� Ȧ��

   // Allocate memory on the device (GPU)
   hipMalloc((void**)&d_A, N * sizeof(float));
   hipMalloc((void**)&d_B, N * sizeof(float));
 
    
   // (1) The array A[N] is copied from CPU to GPU to compute SAXPY.
   hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);

   // Computing A[N] By SAXPY.
   int blocks = (N + THREADS - 1) / THREADS;
   saxpy<<<blocks, THREADS>>>(d_A, d_B, X, Y);

   // (2) The array B[N] is computed by GPU and copied from GPU to CPU.
   hipMemcpy(B, d_B, N * sizeof(float), hipMemcpyDeviceToHost); 
  
   // (3) The array B2[N] is computed by CPU. 
   for(i=0; i<N; i++) {
      B2[i] = A[i]*X + Y; // B2 is used for checking
   }  
    
   // Check the results
   for (i = 0; i < N; i++) 
      if (fabs(B[i] - B2[i]) > 0.001954) // ������ ������ �ִ� �� �����ϴ�. 0.001�� �� i�� 6660���� 0.001���� Ŀ���� ���� ��µǰ� �˴ϴ�. �ּڰ��� 0.001954�� �����Ͽ� �ƹ��͵� ����� �����ʵ��� �����Ͽ����ϴ�. 
         printf("%d: %f %f\n", i, B[i], B2[i]);

   // Free memory on the device
   hipFree(d_A);
   hipFree(d_B);

   return 0; 
}
