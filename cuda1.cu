
#include <hip/hip_runtime.h>
#include <stdio.h>
#define X 12
#define THREAD_X 4

__global__ void index(int *A)
{
   int i = blockDim.x*blockIdx.x+threadIdx.x;
}

int main()
{

   int A[X], *A_d, i;
   
   dim3 dimBlock(THREAD_X);
   dim3 dimGrid(X/dimBlock.x);

   hipSetDevice(1); // or 1
   hipMalloc((void **)&A_d, sizeof(int)*X);
 
   for (i=0; i<X; i++)
      A[i] = -1;
 
   hipMemcpy(A_d, A, sizeof(int)*X, hipMemcpyHostToDevice); 
   
   index<<<dimGrid, dimBlock>>>(A_d);
   
   hipMemcpy(A, A_d, sizeof(int)*X, hipMemcpyDeviceToHost); 
 
   for (i=0; i<X; i++) 
      printf("%d ", A[i]);
   printf("\n");
   
   hipFree(A_d);
}
