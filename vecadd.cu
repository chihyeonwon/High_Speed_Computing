
#include <hip/hip_runtime.h>
#include <stdio.h>
#define THREADS 10
#define N 10000

__global__ void vecAdd(float *A, float*B, float *C)
{
   int i = blockDim.x * blockIdx.x + threadIdx.x;
   if(i < N)
      C[i] = A[i] + B[i];
}

int main()
{
   float A[N], B[N], C[N], C2[N], *A_d, *B_d, *C_d;
   int i;
   
   dim3 dimBlock(THREADS);
   dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x);
   hipSetDevice(1); // or 1

   for (i=0; i<N; i++) {
      A[i] = i*2;
      B[i] = N-i;
      C2[i] = A[i] + B[i]; // C2 is used to check the results
   }


   // allocate array on device
   hipMalloc((void**)&A_d, sizeof(float)*N);
   hipMalloc((void**)&B_d, sizeof(float)*N);
   hipMalloc((void**)&C_d, sizeof(float)*N);


   // copy array from host memory to device memory   
   hipMemcpy(A_d, A, sizeof(float)*N, hipMemcpyHostToDevice);
   hipMemcpy(B_d, B, sizeof(float)*N, hipMemcpyHostToDevice);

   // call GPU function(vecAdd)
   vecAdd<<<dimGrid, dimBlock>>>(A_d, B_d, C_d);

   // Copy the result back
   hipMemcpy(C, C_d, sizeof(float)*N, hipMemcpyDeviceToHost);

   // Check the results
   for (i=0; i<N; i++)
      if (C[i] != C2[i]) printf("%d %f %f\n", i, C[i], C2[i]);

   // free memory on the device
   hipFree(A_d);
   hipFree(B_d);
   hipFree(C_d);

   return 0;
}